
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <time.h>

//Initializing CUDA kernel
//Called from CPU, runs in GPU
__global__ void vector_add(int *a, int *b, int *c, int n)
{
    //calculating globad tid
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    //checking if the tid is not out of bounds
    if(tid<n)
        c[tid] = a[tid] + b[tid];
}

void verify_results(int *a, int *b, int *c, int n)
{
    //Asserting that the results calculated are correct
    for(int i=0; i<n; i++)
    {
        assert(c[i] == a[i] + b[i]);
    }
}

int main()
{
    //Performing operations for 65536 numbers
    int n = 1<<16;    

    //Pointers for CPU vectors  
    int *h_a, *h_b, *h_c;

    //Pointers for GPU vectors
    int *d_a, *d_b, *d_c;

    //Calculate memory needed for each vector
    size_t bytes = n*sizeof(int);

    //Allocate calculated memory on CPU or host
    h_a = (int *) malloc(bytes);
    h_b = (int *) malloc(bytes);
    h_c = (int *) malloc(bytes);

    //Allocate memory on GPU 
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    //Initializing arrays with random numbers
    for (int i=0; i<n; i++)
    {
        h_a[i] = rand();
        h_b[i] = rand();
    }

    //Copying arrays from CPU to GPU
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    //No. of threads per block
    int num_threads = 1024;

    //No. of Thread Blocks
    int num_blocks = (int) ceil(float(n) / num_threads);

    //Starting time to calculate time taken on GPU
    clock_t start = clock();

    //Launch kernel on GPU
    vector_add<<<num_blocks, num_threads>>>(d_a, d_b, d_c, n);

    //Recording end time
    clock_t end = clock();

    //Copying results from GPU to CPU 
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    //Verifying results
    verify_results(h_a, h_b, h_c, n);

    //Free CUDA Memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    //Free CPU Memory
    free(h_a);
    free(h_b);
    free(h_c);

    double time_taken = double(end - start) / CLOCKS_PER_SEC;
    std::cout << "Time Taken on GPU: " << time_taken << std::endl;
    std::cout << "Completed Successfully" << std::endl;

    return 0;
}